#include "hip/hip_runtime.h"
#include "layout.h"
#include <hip/hip_runtime.h>
#include <assert.h>


namespace cuda {

__global__ void cuda_device_init(hiprandState_t *rnd_state_tmp, curandStateCoalesced_t *rnd_state) {
    int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // initialize hiprandState with original hiprand implementation
    hiprand_init(42+tid, tid, 0, &rnd_state_tmp[tid]);
    // copy to coalesced data structure
    rnd_state[blockIdx.x].d[threadIdx.x] = rnd_state_tmp[tid].d;
    rnd_state[blockIdx.x].w0[threadIdx.x] = rnd_state_tmp[tid].v[0];
    rnd_state[blockIdx.x].w1[threadIdx.x] = rnd_state_tmp[tid].v[1];
    rnd_state[blockIdx.x].w2[threadIdx.x] = rnd_state_tmp[tid].v[2];
    rnd_state[blockIdx.x].w3[threadIdx.x] = rnd_state_tmp[tid].v[3];
    rnd_state[blockIdx.x].w4[threadIdx.x] = rnd_state_tmp[tid].v[4];
}

/**
 * @brief: Return 32-bits of pseudorandomness from an XORWOW generator. from "hiprand/hiprand_kernel.h"
 * For some use cases, we don't need floating point uniform distribution. So we don't need to call `curand_uniform_coalesced` as below. We shall use this function. 
 * \param state - Pointer to state to update
 * \param thread_id - Thread id
 * \return 32-bits of pseudorandomness as an unsigned int, all bits valid to use.
*/
__device__ 
unsigned int curand_coalesced(curandStateCoalesced_t *state, uint32_t thread_id) {
    // Return 32-bits of pseudorandomness from an XORWOW generator. 
    uint32_t t;
    t = (state->w0[thread_id] ^ (state->w0[thread_id] >> 2));
    state->w0[thread_id] = state->w1[thread_id];
    state->w1[thread_id] = state->w2[thread_id];
    state->w2[thread_id] = state->w3[thread_id];
    state->w3[thread_id] = state->w4[thread_id];
    state->w4[thread_id] = (state->w4[thread_id] ^ (state->w4[thread_id] << 4)) ^ (t ^ (t << 1));
    state->d[thread_id] += 362437;    
    return state->w4[thread_id] + state->d[thread_id];
}

__device__
float curand_uniform_coalesced(curandStateCoalesced_t *state, uint32_t thread_id) {
    // generate 32 bit pseudorandom value with XORWOW generator (see paper "Xorshift RNGs" by George Marsaglia);
    // also used in hiprand library (see hiprand/hiprand_kernel.h)
    uint32_t t;
    t = state->w0[thread_id] ^ (state->w0[thread_id] >> 2);
    state->w0[thread_id] = state->w1[thread_id];
    state->w1[thread_id] = state->w2[thread_id];
    state->w2[thread_id] = state->w3[thread_id];
    state->w3[thread_id] = state->w4[thread_id];
    state->w4[thread_id] = (state->w4[thread_id] ^ (state->w4[thread_id] << 4)) ^ (t ^ (t << 1));
    state->d[thread_id] += 362437;

    uint32_t rnd_uint = state->d[thread_id] + state->w4[thread_id];

    // convert to float; see hiprand_uniform.h
    return _curand_uniform(rnd_uint);
}


__device__ double compute_zeta(uint32_t n, double theta) {
    double ans = 0.0;
    for (uint32_t i = 1; i <= n; i++) {
        ans += pow(1.0 / double(i), theta);
    }
    return ans;
}

// this function uses the cuda operation __powf, which is a faster but less precise alternative to the pow operation
__device__ uint32_t cuda_rnd_zipf(curandStateCoalesced_t *rnd_state, uint32_t n, double theta, double zeta2, double zetan) {
    double alpha = 1.0 / (1.0 - theta);
    double denominator = 1.0 - zeta2 / zetan;
    if (denominator == 0.0) {
        denominator = 1e-9;
    }
    double eta = (1.0 - __powf(2.0 / double(n), 1.0 - theta)) / (denominator);

    // INFO: hiprand_uniform generates random values between 0.0 (excluded) and 1.0 (included)
    double u = 1.0 - curand_uniform_coalesced(rnd_state, threadIdx.x);
    double uz = u * zetan;

    int64_t val = 0;
    if (uz < 1.0) val = 1;
    else if (uz < 1.0 + __powf(0.5, theta)) val = 2;
    else val = 1 + int64_t(double(n) * __powf(eta * u - eta + 1.0, alpha));

    if (val > n) {
        //printf("WARNING: val: %ld, n: %u\n", val, uint32_t(n));
        val--;
    }
    assert(val >= 0);
    assert(val <= n);
    return uint32_t(val);
}


static __device__ __inline__ uint32_t __mysmid(){
    uint32_t smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    return smid;
}

/**
* @brief: update the coordinates of two visualization nodes in the 2D layout space
* This function is called multiple times in one `cuda_device_layout` in order to increase the data reuse. 
* Each time, the warp shuffle intrinsics are used to change the selection of node 2 among the 32 threads in the warp. 
* E.g. Iter : Step Pairs Selected would be: 
*     1: (a0, b0), (a1, b1), (a2, b2), ..., (a31, b31)
*     2: (a0, b9), (a1, b0), (a2, b3), ..., (a31, b4)
*     3: (a0, b1), (a1, b4), (a2, b1), ..., (a31, b10)
*     ...
* `b` is randomly chosen from the 32 threads in the warp. 
* @param n1_pos_in_path: position of node 1 in the current selected path
* @param n1_id: id of node 1
* @param n1_offset: offset of node 1
* @param n2_pos_in_path: position of node 2 in the current selected path
* @param n2_id: id of node 2
* @param n2_offset: offset of node 2
* @param eta: an coefficient used in the update formula
* @param node_data: the data structure that stores the coordinates of all nodes
*/
__device__
void update_pos_gpu(int64_t &n1_pos_in_path, uint32_t &n1_id, int &n1_offset,
                    int64_t &n2_pos_in_path, uint32_t &n2_id, int &n2_offset,
                    double eta, 
                    cuda::node_data_t &node_data) {
    double term_dist = std::abs(static_cast<double>(n1_pos_in_path) - static_cast<double>(n2_pos_in_path));

    if (term_dist < 1e-9) {
        term_dist = 1e-9;
    }

    double w_ij = 1.0 / term_dist;

    double mu = eta * w_ij;
    if (mu > 1.0) {
        mu = 1.0;
    }

    float *x1 = &node_data.nodes[n1_id].coords[n1_offset];
    float *x2 = &node_data.nodes[n2_id].coords[n2_offset];
    float *y1 = &node_data.nodes[n1_id].coords[n1_offset + 1];
    float *y2 = &node_data.nodes[n2_id].coords[n2_offset + 1];
    double x1_val = double(*x1);
    double x2_val = double(*x2);
    double y1_val = double(*y1);
    double y2_val = double(*y2);

    double dx = x1_val - x2_val;
    double dy = y1_val - y2_val;

    if (dx == 0.0) {
        dx = 1e-9;
    }

    double mag = sqrt(dx * dx + dy * dy);
    double delta = mu * (mag - term_dist) / 2.0;
    //double delta_abs = std::abs(delta);

    // TODO implement delta max stop functionality
    double r = delta / mag;
    double r_x = r * dx;
    double r_y = r * dy;
    // TODO check current value before updating
    atomicExch(x1, float(x1_val - r_x));
    atomicExch(x2, float(x2_val + r_x));
    atomicExch(y1, float(y1_val - r_y));
    atomicExch(y2, float(y2_val + r_y)); 
}

__global__ void cuda_device_layout(int iter, cuda::layout_config_t config, curandStateCoalesced_t *rnd_state, double eta, double *zetas, 
                                   cuda::node_data_t node_data, cuda::path_data_t path_data) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t smid = __mysmid();
    assert(smid < 84);
    curandStateCoalesced_t *thread_rnd_state = &rnd_state[smid];

    __shared__ bool cooling[32];
    if (threadIdx.x % 32 == 1) {
        cooling[threadIdx.x / 32] = (iter >= config.first_cooling_iteration) || (curand_uniform_coalesced(thread_rnd_state, threadIdx.x) <= 0.5);
    }

    // select path
    __shared__ uint32_t first_step_idx[32];
    if (threadIdx.x % 32 == 0) {
        // INFO: hiprand_uniform generates random values between 0.0 (excluded) and 1.0 (included)
        first_step_idx[threadIdx.x / 32] = uint32_t(floor((1.0 - curand_uniform_coalesced(thread_rnd_state, threadIdx.x)) * float(path_data.total_path_steps)));
        assert(first_step_idx[threadIdx.x / 32] < path_data.total_path_steps);
    }
    __syncwarp();

    // find path of step of specific thread with LUT (threads in one warp pick the same path `p`)
    uint32_t step_idx = first_step_idx[threadIdx.x / 32];
    uint32_t path_idx = path_data.element_array[step_idx].pidx;
    path_t p = path_data.paths[path_idx];

    if (p.step_count < 2) {
        return;
    }
    assert(p.step_count > 1);

    // INFO: hiprand_uniform generates random values between 0.0 (excluded) and 1.0 (included)
    uint32_t s1_idx = uint32_t(floor((1.0 - curand_uniform_coalesced(thread_rnd_state, threadIdx.x)) * float(p.step_count)));
    assert(s1_idx < p.step_count);
    uint32_t s2_idx;

    if (cooling[threadIdx.x / 32]) {
        bool backward;
        uint32_t jump_space;
        if (s1_idx > 0 && (curand_uniform_coalesced(thread_rnd_state, threadIdx.x) <= 0.5) || s1_idx == p.step_count-1) {
            // go backward
            backward = true;
            jump_space = min(config.space, s1_idx);
        } else {
            // go forward
            backward = false;
            jump_space = min(config.space, p.step_count - s1_idx - 1);
        }
        uint32_t space = jump_space;
        if (jump_space > config.space_max) {
            space = config.space_max + (jump_space - config.space_max) / config.space_quantization_step + 1;
        }

        uint32_t z_i = cuda_rnd_zipf(thread_rnd_state, jump_space, config.theta, zetas[2], zetas[space]);

        /*
        if (backward) {
            if (!(z_i <= s1_idx)) {
                printf("Error (thread %i): %u - %u\n", threadIdx.x, s1_idx, z_i);
                printf("Jumpspace %u, theta %f, zeta %f\n", jump_space, config.theta, zetas[space]);
            }
            assert(z_i <= s1_idx);
        } else {
            if (!(z_i <= p.step_count - s1_idx - 1)) {
                printf("Error (thread %i): %u + %u, step_count %u\n", threadIdx.x, s1_idx, z_i, p.step_count);
                printf("Jumpspace %u, theta %f, zeta %f\n", jump_space, config.theta, zetas[space]);
            }
            assert(s1_idx + z_i < p.step_count);
        }
        */

        s2_idx = backward? s1_idx - z_i: s1_idx + z_i;
    } else {
        do {
            s2_idx = uint32_t(floor((1.0 - curand_uniform_coalesced(thread_rnd_state, threadIdx.x)) * float(p.step_count)));
        } while (s1_idx == s2_idx);
    }
    assert(s1_idx < p.step_count);
    assert(s2_idx < p.step_count);
    assert(s1_idx != s2_idx);


    uint32_t n1_id = p.elements[s1_idx].node_id;
    int64_t n1_pos_in_path = p.elements[s1_idx].pos;
    bool n1_is_rev = (n1_pos_in_path < 0)? true: false;
    n1_pos_in_path = std::abs(n1_pos_in_path);

    uint32_t n2_id = p.elements[s2_idx].node_id;
    int64_t n2_pos_in_path = p.elements[s2_idx].pos;
    bool n2_is_rev = (n2_pos_in_path < 0)? true: false;
    n2_pos_in_path = std::abs(n2_pos_in_path);

    uint32_t n1_seq_length = node_data.nodes[n1_id].seq_length;
    bool n1_use_other_end = (curand_uniform_coalesced(thread_rnd_state, threadIdx.x) <= 0.5)? true: false;
    if (n1_use_other_end) {
        n1_pos_in_path += uint64_t{n1_seq_length};
        n1_use_other_end = !n1_is_rev;
    } else {
        n1_use_other_end = n1_is_rev;
    }

    uint32_t n2_seq_length = node_data.nodes[n2_id].seq_length;
    bool n2_use_other_end = (curand_uniform_coalesced(thread_rnd_state, threadIdx.x) <= 0.5)? true: false;
    if (n2_use_other_end) {
        n2_pos_in_path += uint64_t{n2_seq_length};
        n2_use_other_end = !n2_is_rev;
    } else {
        n2_use_other_end = n2_is_rev;
    }

    int n1_offset = n1_use_other_end? 2: 0;
    int n2_offset = n2_use_other_end? 2: 0;

    // Update Coordinates based on the data of selected nodes: n_pos_in_path, n_id, n_offset
    update_pos_gpu(n1_pos_in_path, n1_id, n1_offset, 
                   n2_pos_in_path, n2_id, n2_offset, 
                   eta, node_data);

#define UPDATE_TIMES 2
    // Data Reuse for the non-cooling iteration
    if (!cooling[threadIdx.x / 32]) {
        // Shuffle and Update (DATA_REUSE_TIMES = UPDATE_TIMES - 1) times (UPDATE_TIMES is the total update times when calling `cuda_device_layout` once)
        for (int i = 0; i < UPDATE_TIMES - 1; i++) {
            // Shuffle the step data within a warp
            int shuffle_laneId = curand_coalesced(thread_rnd_state, threadIdx.x) % 32;
            uint64_t n2_pos_in_path_tmp = __shfl_sync(0xffffffff, n2_pos_in_path, shuffle_laneId);
            uint32_t n2_id_tmp = __shfl_sync(0xffffffff, n2_id, shuffle_laneId);
            int n2_offset_tmp = __shfl_sync(0xffffffff, n2_offset, shuffle_laneId);

            n2_pos_in_path = n2_pos_in_path_tmp;
            n2_id = n2_id_tmp;
            n2_offset = n2_offset_tmp;

            if ((n1_id != n2_id) || (n1_offset != n2_offset)) { // Only update if the two nodes are different
                update_pos_gpu(n1_pos_in_path, n1_id, n1_offset,
                            n2_pos_in_path, n2_id, n2_offset,
                            eta, node_data);
            }

        }
    }

}


void cpu_layout(cuda::layout_config_t config, double *etas, double *zetas, cuda::node_data_t &node_data, cuda::path_data_t &path_data) {
    int nbr_threads = config.nthreads;
    std::cout << "cuda cpu layout (" << nbr_threads << " threads)" << std::endl;
    std::vector<uint64_t> path_dist;
    for (int p = 0; p < path_data.path_count; p++) {
        path_dist.push_back(uint64_t(path_data.paths[p].step_count));
    }

#pragma omp parallel num_threads(nbr_threads)
    {
        int tid = omp_get_thread_num();

        XoshiroCpp::Xoshiro256Plus gen(9399220 + tid);
        std::uniform_int_distribution<uint64_t> flip(0, 1);
        std::discrete_distribution<> rand_path(path_dist.begin(), path_dist.end());

        const int steps_per_thread = config.min_term_updates / nbr_threads;

//#define profiling
#ifdef profiling
        auto total_duration_dist = std::chrono::duration<double>::zero(); // total time on computing distance: in seconds
        auto total_duration_sgd = std::chrono::duration<double>::zero(); // total time on SGD: in seconds
        // detailed analysis on different parts of Updating Coordinates Part
        auto total_duration_compute_first = std::chrono::duration<double>::zero();
        auto total_duration_load = std::chrono::duration<double>::zero();
        auto total_duration_compute_second = std::chrono::duration<double>::zero();
        auto total_duration_store = std::chrono::duration<double>::zero();
        // detailed analysis on different parts of Getting Distance Part
        auto total_duration_one_step_gen = std::chrono::duration<double>::zero();
        auto total_duration_two_step_gen = std::chrono::duration<double>::zero();
        auto total_duration_get_distance = std::chrono::duration<double>::zero();


        std::chrono::high_resolution_clock::time_point start_dist;
        std::chrono::high_resolution_clock::time_point end_dist;
        std::chrono::high_resolution_clock::time_point start_sgd;
        std::chrono::high_resolution_clock::time_point one_step_gen;
        std::chrono::high_resolution_clock::time_point two_step_gen;

        // detailed analysis on Updating Coordinates part
        std::chrono::high_resolution_clock::time_point before_load;
        std::chrono::high_resolution_clock::time_point after_load;
        std::chrono::high_resolution_clock::time_point before_store;
        std::chrono::high_resolution_clock::time_point after_store;
#endif

        for (int iter = 0; iter < config.iter_max; iter++ ) {
            // synchronize all threads before each iteration
#pragma omp barrier
            for (int step = 0; step < steps_per_thread; step++ ) {
#ifdef profiling
                start_dist = std::chrono::high_resolution_clock::now();
#endif
                // get path
                uint32_t path_idx = rand_path(gen);
                path_t p = path_data.paths[path_idx];
                if (p.step_count < 2) {
                    continue;
                }

                std::uniform_int_distribution<uint32_t> rand_step(0, p.step_count-1);

                uint32_t s1_idx = rand_step(gen);
#ifdef profiling
                one_step_gen = std::chrono::high_resolution_clock::now();
                total_duration_one_step_gen += std::chrono::duration_cast<std::chrono::nanoseconds>(one_step_gen - start_dist);
#endif
                uint32_t s2_idx;
                if (iter >= config.first_cooling_iteration || flip(gen)) {
                    if (s1_idx > 0 && flip(gen) || s1_idx == p.step_count-1) {
                        // go backward
                        uint32_t jump_space = std::min(config.space, s1_idx);
                        uint32_t space = jump_space;
                        if (jump_space > config.space_max) {
                            space = config.space_max + (jump_space - config.space_max) / config.space_quantization_step + 1;
                        }
                        dirtyzipf::dirty_zipfian_int_distribution<uint64_t>::param_type z_p(1, jump_space, config.theta, zetas[space]);
                        dirtyzipf::dirty_zipfian_int_distribution<uint64_t> z(z_p);
                        uint32_t z_i = (uint32_t) z(gen);
                        s2_idx = s1_idx - z_i;
                    } else {
                        // go forward
                        uint32_t jump_space = std::min(config.space, p.step_count - s1_idx - 1);
                        uint32_t space = jump_space;
                        if (jump_space > config.space_max) {
                            space = config.space_max + (jump_space - config.space_max) / config.space_quantization_step + 1;
                        }
                        dirtyzipf::dirty_zipfian_int_distribution<uint64_t>::param_type z_p(1, jump_space, config.theta, zetas[space]);
                        dirtyzipf::dirty_zipfian_int_distribution<uint64_t> z(z_p);
                        uint32_t z_i = (uint32_t) z(gen);
                        s2_idx = s1_idx + z_i;
                    }
                } else {
                    do {
                        s2_idx = rand_step(gen);
                    } while (s1_idx == s2_idx);
                }
#ifdef profiling
                two_step_gen = std::chrono::high_resolution_clock::now();
                total_duration_two_step_gen += std::chrono::duration_cast<std::chrono::nanoseconds>(two_step_gen - one_step_gen);
#endif
                assert(s1_idx < p.step_count);
                assert(s2_idx < p.step_count);

                uint32_t n1_id = p.elements[s1_idx].node_id;
                int64_t n1_pos_in_path = p.elements[s1_idx].pos;
                bool n1_is_rev = (n1_pos_in_path < 0)? true: false;
                n1_pos_in_path = std::abs(n1_pos_in_path);

                uint32_t n2_id = p.elements[s2_idx].node_id;
                int64_t n2_pos_in_path = p.elements[s2_idx].pos;
                bool n2_is_rev = (n2_pos_in_path < 0)? true: false;
                n2_pos_in_path = std::abs(n2_pos_in_path);

                uint32_t n1_seq_length = node_data.nodes[n1_id].seq_length;
                bool n1_use_other_end = flip(gen);
                if (n1_use_other_end) {
                    n1_pos_in_path += uint64_t{n1_seq_length};
                    n1_use_other_end = !n1_is_rev;
                } else {
                    n1_use_other_end = n1_is_rev;
                }

                uint32_t n2_seq_length = node_data.nodes[n2_id].seq_length;
                bool n2_use_other_end = flip(gen);
                if (n2_use_other_end) {
                    n2_pos_in_path += uint64_t{n2_seq_length};
                    n2_use_other_end = !n2_is_rev;
                } else {
                    n2_use_other_end = n2_is_rev;
                }

                double term_dist = std::abs(static_cast<double>(n1_pos_in_path) - static_cast<double>(n2_pos_in_path));

                if (term_dist < 1e-9) {
                    term_dist = 1e-9;
                }
#ifdef profiling
                end_dist = std::chrono::high_resolution_clock::now();
                total_duration_get_distance += std::chrono::duration_cast<std::chrono::nanoseconds>(end_dist - two_step_gen);

                total_duration_dist += std::chrono::duration_cast<std::chrono::nanoseconds>(end_dist - start_dist);

                start_sgd = std::chrono::high_resolution_clock::now();
#endif

                double w_ij = 1.0 / term_dist;

                double mu = etas[iter] * w_ij;
                if (mu > 1.0) {
                    mu = 1.0;
                }

                double d_ij = term_dist;

                int n1_offset = n1_use_other_end? 2: 0;
                int n2_offset = n2_use_other_end? 2: 0;

#ifdef profiling
                before_load = std::chrono::high_resolution_clock::now();
                total_duration_compute_first += std::chrono::duration_cast<std::chrono::nanoseconds>(before_load - start_sgd);
#endif
                float *x1 = &node_data.nodes[n1_id].coords[n1_offset];
                float *x2 = &node_data.nodes[n2_id].coords[n2_offset];
                float *y1 = &node_data.nodes[n1_id].coords[n1_offset + 1];
                float *y2 = &node_data.nodes[n2_id].coords[n2_offset + 1];

                double dx = float(*x1 - *x2);
                double dy = float(*y1 - *y2);
#ifdef profiling
                after_load = std::chrono::high_resolution_clock::now();
                total_duration_load += std::chrono::duration_cast<std::chrono::nanoseconds>(after_load - before_load);
#endif
                if (dx == 0.0) {
                    dx = 1e-9;
                }

                double mag = sqrt(dx * dx + dy * dy);
                double delta = mu * (mag - d_ij) / 2.0;
                //double delta_abs = std::abs(delta);

                double r = delta / mag;
                double r_x = r * dx;
                double r_y = r * dy;

#ifdef profiling
                before_store = std::chrono::high_resolution_clock::now();
                total_duration_compute_second += std::chrono::duration_cast<std::chrono::nanoseconds>(before_store - after_load);
#endif
                *x1 -= float(r_x);
                *y1 -= float(r_y);
                *x2 += float(r_x);
                *y2 += float(r_y);
#ifdef profiling
                after_store = std::chrono::high_resolution_clock::now();
                total_duration_store += std::chrono::duration_cast<std::chrono::nanoseconds>(after_store - before_store);
                total_duration_sgd += std::chrono::duration_cast<std::chrono::nanoseconds>(after_store - start_sgd);
#endif
            }
        }

#ifdef profiling
        std::stringstream msg;
        msg << "Thread[" << tid << "]: Dataloading time = " << total_duration_dist.count() << " sec;\t" << "Compute time = " << total_duration_sgd.count() << " sec." << std::endl;

        msg << std::left
            << std::setw(40) << "Getting Distance Part Breakdown: " << std::endl
            << std::setw(20) << "[0] One Step Gen: "
            << std::setw(10) << total_duration_one_step_gen.count()
            << std::setw(10)  << " sec;"
            << std::setw(20) << "[1] Two Steps Gen: "
            << std::setw(10) << total_duration_two_step_gen.count()
            << std::setw(10)  << " sec;"
            << std::setw(20) << "[2] Get Distance: "
            << std::setw(10) << total_duration_get_distance.count()
            << std::setw(10) << " sec."
            << std::endl;

        msg << std::setw(40) << "Updating Coordinate Part Breakdown: " << std::endl
            << std::setw(20) << "[0] First Compute: "
            << std::setw(10) << total_duration_compute_first.count()
            << std::setw(10)  << " sec;"
            << std::setw(20) << "[1] Load Pos: "
            << std::setw(10) << total_duration_load.count()
            << std::setw(10)  << " sec;"
            << std::setw(20) << "[2] Second Compute: "
            << std::setw(10) << total_duration_compute_second.count()
            << std::setw(10)  << " sec;"
            << std::setw(20) << "[3] Update Pos: "
            << std::setw(10) << total_duration_store.count()
            << std::setw(10)  << " sec."
            << std::endl << std::endl;

        std::cerr << msg.str();
#endif

    }
}


void cuda_layout(layout_config_t config, const odgi::graph_t &graph, std::vector<std::atomic<double>> &X, std::vector<std::atomic<double>> &Y) {

#ifdef cuda_layout_profiling
    auto start = std::chrono::high_resolution_clock::now();
#endif


    std::cout << "Hello world from CUDA host" << std::endl;
    std::cout << "iter_max: " << config.iter_max << std::endl;
    std::cout << "first_cooling_iteration: " << config.first_cooling_iteration << std::endl;
    std::cout << "min_term_updates: " << config.min_term_updates << std::endl;
    std::cout << "size of node_t: " << sizeof(node_t) << std::endl;
    std::cout << "theta: " << config.theta << std::endl;

    // create eta array
    double *etas;
    hipMallocManaged(&etas, config.iter_max * sizeof(double));

    const int32_t iter_max = config.iter_max;
    const int32_t iter_with_max_learning_rate = config.iter_with_max_learning_rate;
    const double w_max = 1.0;
    const double eps = config.eps;
    const double eta_max = config.eta_max;
    const double eta_min = eps / w_max;
    const double lambda = log(eta_max / eta_min) / ((double) iter_max - 1);
    for (int32_t i = 0; i < config.iter_max; i++) {
        double eta = eta_max * exp(-lambda * (std::abs(i - iter_with_max_learning_rate)));
        etas[i] = isnan(eta)? eta_min : eta;
    }


    // create node data structure
    // consisting of sequence length and coords
    uint32_t node_count = graph.get_node_count();
    std::cout << "node_count: " << node_count << std::endl;
    assert(graph.min_node_id() == 1);
    assert(graph.max_node_id() == node_count);
    assert(graph.max_node_id() - graph.min_node_id() + 1 == node_count);

    cuda::node_data_t node_data;
    node_data.node_count = node_count;
    hipMallocManaged(&node_data.nodes, node_count * sizeof(cuda::node_t));
    for (int node_idx = 0; node_idx < node_count; node_idx++) {
        //assert(graph.has_node(node_idx));
        cuda::node_t *n_tmp = &node_data.nodes[node_idx];

        // sequence length
        const handlegraph::handle_t h = graph.get_handle(node_idx + 1, false);
        // NOTE: unable store orientation (reverse), since this information is path dependent
        n_tmp->seq_length = graph.get_length(h);

        // copy random coordinates
        n_tmp->coords[0] = float(X[node_idx * 2].load());
        n_tmp->coords[1] = float(Y[node_idx * 2].load());
        n_tmp->coords[2] = float(X[node_idx * 2 + 1].load());
        n_tmp->coords[3] = float(Y[node_idx * 2 + 1].load());
    }


    // create path data structure
    uint32_t path_count = graph.get_path_count();
    cuda::path_data_t path_data;
    path_data.path_count = path_count;
    path_data.total_path_steps = 0;
    hipMallocManaged(&path_data.paths, path_count * sizeof(cuda::path_t));

    vector<odgi::path_handle_t> path_handles{};
    path_handles.reserve(path_count);
    graph.for_each_path_handle(
        [&] (const odgi::path_handle_t& p) {
            path_handles.push_back(p);
            path_data.total_path_steps += graph.get_step_count(p);
        });
    hipMallocManaged(&path_data.element_array, path_data.total_path_steps * sizeof(path_element_t));

    // get length and starting position of all paths
    uint32_t first_step_counter = 0;
    for (int path_idx = 0; path_idx < path_count; path_idx++) {
        odgi::path_handle_t p = path_handles[path_idx];
        int step_count = graph.get_step_count(p);
        path_data.paths[path_idx].step_count = step_count;
        path_data.paths[path_idx].first_step_in_path = first_step_counter;
        first_step_counter += step_count;
    }

#pragma omp parallel for num_threads(config.nthreads)
    for (int path_idx = 0; path_idx < path_count; path_idx++) {
        odgi::path_handle_t p = path_handles[path_idx];
        //std::cout << graph.get_path_name(p) << ": " << graph.get_step_count(p) << std::endl;

        uint32_t step_count = path_data.paths[path_idx].step_count;
        uint32_t first_step_in_path = path_data.paths[path_idx].first_step_in_path;
        if (step_count == 0) {
            path_data.paths[path_idx].elements = NULL;
        } else {
            path_element_t *cur_path = &path_data.element_array[first_step_in_path];
            path_data.paths[path_idx].elements = cur_path;

            odgi::step_handle_t s = graph.path_begin(p);
            int64_t pos = 1;
            // Iterate through path
            for (int step_idx = 0; step_idx < step_count; step_idx++) {
                odgi::handle_t h = graph.get_handle_of_step(s);
                //std::cout << graph.get_id(h) << std::endl;

                cur_path[step_idx].node_id = graph.get_id(h) - 1;
                cur_path[step_idx].pidx = uint32_t(path_idx);
                // store position negative when handle reverse
                if (graph.get_is_reverse(h)) {
                    cur_path[step_idx].pos = -pos;
                } else {
                    cur_path[step_idx].pos = pos;
                }
                pos += graph.get_length(h);

                // get next step
                if (graph.has_next_step(s)) {
                    s = graph.get_next_step(s);
                } else if (!(step_idx == step_count-1)) {
                    // should never be reached
                    std::cout << "Error: Here should be another step" << std::endl;
                }
            }
        }
    }


    // cache zipf zetas
    auto start_zeta = std::chrono::high_resolution_clock::now();
    double *zetas;
    uint64_t zetas_cnt = ((config.space <= config.space_max)? config.space : (config.space_max + (config.space - config.space_max) / config.space_quantization_step + 1)) + 1;
    std::cout << "zetas_cnt: " << zetas_cnt << std::endl;
    std::cout << "space_max: " << config.space_max << std::endl;
    std::cout << "config.space: " << config.space << std::endl;
    std::cout << "config.space_quantization: " << config.space_quantization_step << std::endl;

    hipMallocManaged(&zetas, zetas_cnt * sizeof(double));
    double zeta_tmp = 0.0;
    for (uint64_t i = 1; i < config.space + 1; i++) {
        zeta_tmp += dirtyzipf::fast_precise_pow(1.0 / i, config.theta);
        if (i <= config.space_max) {
            zetas[i] = zeta_tmp;
        }
        if (i >= config.space_max && (i - config.space_max) % config.space_quantization_step == 0) {
            zetas[config.space_max + 1 + (i - config.space_max) / config.space_quantization_step] = zeta_tmp;
        }
    }
    auto end_zeta = std::chrono::high_resolution_clock::now();
    uint32_t duration_zeta_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_zeta - start_zeta).count();
    std::cout << "Zeta precompute took " << duration_zeta_ms << "ms" << std::endl;


    auto start_compute = std::chrono::high_resolution_clock::now();
#define USE_GPU
#ifdef USE_GPU
    std::cout << "cuda gpu layout" << std::endl;
    std::cout << "total-path_steps: " << path_data.total_path_steps << std::endl;

    const uint64_t block_size = BLOCK_SIZE;
    uint64_t block_nbr = (config.min_term_updates + block_size - 1) / block_size;

#define STEP_DECREASE_FACTOR 1.75
    // block_nbr = block_nbr / STEP_DECREASE_FACTOR; but note the type conversion
    block_nbr = uint64_t(double(block_nbr) / STEP_DECREASE_FACTOR);   

    std::cout << "block_nbr: " << block_nbr << " block_size: " << block_size << std::endl;
    hiprandState_t *rnd_state_tmp;
    curandStateCoalesced_t *rnd_state;
    hipError_t tmp_error = hipMallocManaged(&rnd_state_tmp, SM_COUNT * block_size * sizeof(hiprandState_t));
    std::cout << "rnd state CUDA Error: " << hipGetErrorName(tmp_error) << ": " << hipGetErrorString(tmp_error) << std::endl;
    tmp_error = hipMallocManaged(&rnd_state, SM_COUNT * sizeof(curandStateCoalesced_t));
    std::cout << "rnd state CUDA Error: " << hipGetErrorName(tmp_error) << ": " << hipGetErrorString(tmp_error) << std::endl;
    cuda_device_init<<<SM_COUNT, block_size>>>(rnd_state_tmp, rnd_state);
    tmp_error = hipDeviceSynchronize();
    std::cout << "rnd state CUDA Error: " << hipGetErrorName(tmp_error) << ": " << hipGetErrorString(tmp_error) << std::endl;
    hipFree(rnd_state_tmp);


    for (int iter = 0; iter < config.iter_max; iter++) {
        cuda_device_layout<<<block_nbr, block_size>>>(iter, config, rnd_state, etas[iter], zetas, node_data, path_data);
        hipError_t error = hipDeviceSynchronize();
        std::cout << "CUDA Error: " << hipGetErrorName(error) << ": " << hipGetErrorString(error) << std::endl;
    }

#else
    cpu_layout(config, etas, zetas, node_data, path_data);
#endif
    auto end_compute = std::chrono::high_resolution_clock::now();
    uint32_t duration_compute_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_compute - start_compute).count();
    std::cout << "CUDA layout compute took " << duration_compute_ms << "ms" << std::endl;



    // copy coords back to X, Y vectors
    for (int node_idx = 0; node_idx < node_count; node_idx++) {
        cuda::node_t *n = &(node_data.nodes[node_idx]);
        // coords[0], coords[1], coords[2], coords[3] are stored consecutively. 
        float *coords = n->coords;
        // check if coordinates valid (not NaN or infinite)
        for (int i = 0; i < 4; i++) {
            if (!isfinite(coords[i])) {
                std::cout << "WARNING: invalid coordiate" << std::endl;
            }
        }
        X[node_idx * 2].store(double(coords[0]));
        Y[node_idx * 2].store(double(coords[1]));
        X[node_idx * 2 + 1].store(double(coords[2]));
        Y[node_idx * 2 + 1].store(double(coords[3]));
        //std::cout << "coords of " << node_idx << ": [" << X[node_idx*2] << "; " << Y[node_idx*2] << "] ; [" << X[node_idx*2+1] << "; " << Y[node_idx*2+1] <<"]\n";
    }


    // get rid of CUDA data structures
    hipFree(etas);
    hipFree(node_data.nodes);
    hipFree(path_data.paths);
    hipFree(path_data.element_array);
    hipFree(zetas);
#ifdef USE_GPU
    hipFree(rnd_state);
#endif


#ifdef cuda_layout_profiling
    auto end = std::chrono::high_resolution_clock::now();
    uint32_t duration_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "CUDA layout took " << duration_ms << "ms" << std::endl;
#endif

    return;
}

}
